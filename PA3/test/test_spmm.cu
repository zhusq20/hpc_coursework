#include "gtest/gtest.h"
#include "util.h"
#include "valid.h"
#include "spmm_ref.h"
#include "spmm_opt.h"
#include "spmm_cusparse.h"

class SpMMTest : public testing::Test
{
protected:
    vector<void *> tensor_ptr;
    float *p_in_feat_vec, *p_out_feat_vec, *p_out_feat_vec_ref, *p_value, *p_value2;
    int *gptr2, *gidx2;
    CSR *g, *g2;
    virtual void SetUp()
    {
        p_in_feat_vec = allocate<float>(kNumV * kLen, &tensor_ptr);
        p_out_feat_vec = allocate<float>(kNumV * kLen, &tensor_ptr);
        p_out_feat_vec_ref = allocate<float>(kNumV * kLen, &tensor_ptr);
        p_value = allocate<float>(kNumE, &tensor_ptr);
	
	gptr2 = allocate<int>(kNumV + 1, &tensor_ptr);
	gidx2 = allocate<int>(kNumE, &tensor_ptr);
	p_value2 = allocate<float>(kNumE, &tensor_ptr);
	checkCudaErrors(hipMemcpy(gptr2, gptr, (kNumV + 1) * sizeof(int), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(gidx2, gidx, kNumE * sizeof(int), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(p_value2, p_value, kNumE * sizeof(float), hipMemcpyDeviceToDevice));

        g = new CSR(kNumV, kNumE, gptr, gidx, p_value);
	g2 = new CSR(kNumV, kNumE, gptr2, gidx2, p_value2);
    }
    virtual void TearDown()
    {
        for (auto item : tensor_ptr)
        {
            hipFree(item);
        }
    }
};

TEST_F(SpMMTest, validation)
{
    SpMMRef *spmmer_ref = new SpMMRef(g, kLen);
    SpMMOpt *spmmer = new SpMMOpt(g2, kLen);
    spmmer_ref->preprocess(p_in_feat_vec, p_out_feat_vec_ref);
    spmmer->preprocess(p_in_feat_vec, p_out_feat_vec);
    checkCudaErrors(hipMemset(p_out_feat_vec, 0, sizeof(float) * kNumV * kLen));
    checkCudaErrors(hipMemset(p_out_feat_vec_ref, 0, sizeof(float) * kNumV * kLen));
    spmmer_ref->run(p_in_feat_vec, p_out_feat_vec_ref);
    spmmer->run(p_in_feat_vec, p_out_feat_vec);
    checkCudaErrors(hipDeviceSynchronize());
    // This ASSERT will fail because your SpMM is not implemented yet
    ASSERT_LT(valid(p_out_feat_vec, p_out_feat_vec_ref, kNumV * kLen), kNumV * kLen / 10000 + 1);
}

TEST_F(SpMMTest, cusparse_performance)
{
    SpMMCuSparse *spmmer = new SpMMCuSparse(g, kLen);
    spmmer->preprocess(p_in_feat_vec, p_out_feat_vec);
    auto time = getAverageTimeWithWarmUp([&]()
                                         { spmmer->run(p_in_feat_vec, p_out_feat_vec); });
    dbg(time);
}

TEST_F(SpMMTest, opt_performance)
{
    SpMMOpt *spmmer = new SpMMOpt(g, kLen);
    spmmer->preprocess(p_in_feat_vec, p_out_feat_vec);
    auto time = getAverageTimeWithWarmUp([&]()
                                         { spmmer->run(p_in_feat_vec, p_out_feat_vec); });
    dbg(time);
}
